#include <stdio.h>
#include <assert.h>
#include <hipcub/hipcub.hpp>
#include "radix_sort.h"


using namespace hipcub;

float radix_sort(int *keys, Particle *ps, int offset, int num_items)
{ 
  /* Note that keys and ps should be device pointers! */

  hipError_t err; 
  hipEvent_t start, end;
  int *keys_alt;
  Particle *ps_alt;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventCreate(&end);

  // allocate key and value double buffers on the device
  err = hipMalloc((void**) &keys_alt, num_items*sizeof(int));
  assert(err==0);

  err = hipMalloc((void**) &ps_alt, num_items*sizeof(Particle));
  assert(err==0);
  
  printf("offset = %d\n",offset);

  hipcub::DoubleBuffer<int> d_keys(keys+offset, keys_alt);
  hipcub::DoubleBuffer<Particle> d_vals(ps+offset, ps_alt);

  // Determine temporary device storage requirements for sorting operation
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items);
  // Allocate temporary storage for sorting operation
  err = hipMalloc(&d_temp_storage, temp_storage_bytes);
  assert(err==0);
 
  // Run sorting operation
  hipEventRecord(start,0);
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items);
  hipEventRecord(end,0);
  hipEventSynchronize(end);
  hipDeviceSynchronize();

  hipEventElapsedTime(&elapsedTime,start,end);
  printf("Sort time on GPU = %f ms, %f million keys/s\n", elapsedTime, (float)num_items/elapsedTime*1e3/1e6);

  //Sorted keys are referenced by d_keys.Current()
  keys = d_keys.Current();
  ps = d_vals.Current();
    
  
  hipFree(keys_alt);
  hipFree(ps_alt);
  hipFree(d_temp_storage);
  return elapsedTime;
}
